#include "hip/hip_runtime.h"
#include <vector>
#include <stack>
#include <cmath>
#include <cassert>
#include <algorithm>

#include <hip/hip_runtime.h>

#include "brute_force.h"
#include "tsp.h"

__global__ void solveTSPGPUKernel(dist_t *dists, int *optimPath, int *currentPath, dist_t *optimPathLen, size_t pathSize, size_t iterationsNumber, int lastTownNumber)
{
    // TODO: use shared memory

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int threadsNumber = blockDim.x * gridDim.x;
    size_t iters = iterationsNumber / threadsNumber;
    size_t startIter = idx * iters;
    if (idx == threadsNumber - 1)
    {
        // For last thread we take also the rest of iterations
        iters += iterationsNumber % threadsNumber;
    }

    size_t townsNumberToIterate = static_cast<size_t>(lastTownNumber);
    size_t currentIter = startIter;

    for (long i = pathSize - 1; i >= 0; --i)
    {
        // On the (currentIter == 1) current path should be 1 1 ... 1 2
        // On the (currentIter == townsNumberToIterate) it should be check of 1 1 ... 1 2 1
        currentPath[idx * pathSize + i] = static_cast<int>(currentIter % townsNumberToIterate + 1);
        currentIter = currentIter / townsNumberToIterate;
    }

    optimPathLen[idx] = 123456;

    for (size_t i = 0; i < iters; ++i)
    {
        // Check that current path is valid
        bool skip = false;
        for (size_t j1 = 0; j1 < pathSize; ++j1)
        {
            for (size_t j2 = 0; j2 < pathSize; ++j2)
            {
                if (j1 != j2 && currentPath[idx * pathSize + j1] == currentPath[idx * pathSize + j2])
                {
                    skip = true;
                    break;
                }
            }
            if (skip)
                break;
        }

        // Compute current path
        if (!skip)
        {
            int previousTown = 0;
            dist_t currentPathLen = 0;
            int townsTotalNumber = lastTownNumber + 1;

            for (size_t j = 0; j < pathSize; ++j)
            {
                int town = currentPath[idx * pathSize + j];
                currentPathLen += dists[previousTown * townsTotalNumber + town];
                previousTown = town;
            }
            currentPathLen += dists[previousTown * townsTotalNumber]; // plus distance to the first town (0)

            if (optimPathLen[idx] == 0 || optimPathLen[idx] > currentPathLen)
            {
                optimPathLen[idx] = currentPathLen;
                // std::copy(currentPath + idx * pathSize, currentPath + idx * pathSize + pathSize, optimPath + idx * pathSize);
                for (size_t k = 0; k < pathSize; ++k)
                {
                    (optimPath + idx * pathSize)[i] = (currentPath + idx * pathSize)[i];
                }
            }
        }

        // Set next path
        // lastNonLastIndex: Index of the last town in a path which is not equal to lastTownNumber (e.g. if lastTownNumber == 5, path == [2 1 5 3 2] => lastNonLastIndex == 2)
        size_t lastNonLastIndex = pathSize - 1;

        while (currentPath[idx * pathSize + lastNonLastIndex] == lastTownNumber)
        {
            currentPath[idx * pathSize + lastNonLastIndex] = 1;
            assert(lastNonLastIndex > 0 || (idx == threadsNumber - 1 && i == iters));
            lastNonLastIndex--;
        }

        currentPath[idx * pathSize + lastNonLastIndex]++;
    }
}

namespace BruteForce
{
    void solveTSPGPU(TSP &tsp, size_t threadsNumber)
    {
        const size_t townsNumber = tsp.getTownsNumber();
        size_t pathSize = townsNumber - 1; // not include last move to the start town
        int lastTownNumber = static_cast<int>(townsNumber - 1);

        // we will place towns with numbers [1-lastTownNumber] on pathSize places
        size_t iterationsNumber = std::pow(townsNumber - 1, pathSize);
        std::cout << "Consts: " << townsNumber << " " << pathSize << " " << lastTownNumber << " " << iterationsNumber << " " << std::endl;

        dist_t *d_optimPathLen;
        int *d_optimPath;
        int *d_currentPath;
        dist_t *d_dists;
        hipMalloc((void **)&d_optimPathLen, threadsNumber * sizeof(dist_t));
        hipMalloc((void **)&d_optimPath, threadsNumber * pathSize * sizeof(int));
        hipMalloc((void **)&d_currentPath, threadsNumber * pathSize * sizeof(int));
        hipMalloc((void **)&d_dists, townsNumber * townsNumber * sizeof(int));

        hipMemcpy(d_dists, tsp.getDists(), townsNumber * townsNumber * sizeof(dist_t), hipMemcpyHostToDevice);
        std::cout << "tsp.getDists()[2]: " << tsp.getDists()[2] << std::endl;

        solveTSPGPUKernel<<<1, threadsNumber>>>(d_dists, d_optimPath, d_currentPath, d_optimPathLen, pathSize, iterationsNumber, lastTownNumber);

        dist_t *h_optimPathLen = new dist_t[threadsNumber];
        hipMemcpy(h_optimPathLen, d_optimPathLen, threadsNumber * sizeof(dist_t), hipMemcpyDeviceToHost);
        std::cout << "h_optimPathLen: " << std::endl;
        for (int i = 0; i < threadsNumber; ++i)
        {
            std::cout << h_optimPathLen[i] << " ";
        }
        std::cout << std::endl;

        dist_t *argMinOptimPathLenIter = std::min_element(h_optimPathLen, h_optimPathLen + threadsNumber);
        size_t argMinOptimPathLen = argMinOptimPathLenIter - h_optimPathLen;
        assert(argMinOptimPathLen < threadsNumber);
        dist_t optimPathLen = h_optimPathLen[argMinOptimPathLen];

        int *optimPath = new int[pathSize];
        hipMemcpy(optimPath, d_optimPath + argMinOptimPathLen * pathSize, pathSize * sizeof(int), hipMemcpyDeviceToHost);
        std::cout << "Optim path before set: " << optimPathLen << std::endl;
        for (int i = 0; i < pathSize; ++i)
        {
            std::cout << optimPath[i] << " ";
        }
        std::cout << std::endl;

        tsp.setSolution(optimPath, optimPathLen);
    }
}